#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


/*
 * Notice the absence of the previously expected argument `N`.
 */

__global__ void loop(int N)
{
  /*
   * This kernel does the work of only 1 iteration
   * of the original for loop. Indication of which
   * "iteration" is being executed by this kernel is
   * still available via `threadIdx.x`.
   */
  int i = blockIdx.x*blockdim.x+threadId.x;
  printf("This is iteration number %d\n", threadIdx.x);
}

int main()
{
  /*
   * It is the execution context that sets how many "iterations"
   * of the "loop" will be done.
   */
  int N=10;
  loop<<<10,1>>>(N);

  hipDeviceSynchronize();
}
