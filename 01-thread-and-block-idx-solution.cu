#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void printSuccessForCorrectExecutionConfiguration()
{

  if(threadIdx.x == 1023 && blockIdx.x == 255)
  {
    printf("Success!\n");
  } else {
    printf("Failure. Update the execution configuration as necessary.\n")
  }
}

int main()
{
  /*
   * This is one possible execution context that will make
   * the kernel launch print its success message.
   */

  printSuccessForCorrectExecutionConfiguration<<<250, 1024>>>();

  /*
   * Don't forget kernel execution is asynchronous and you must
   * sync on its completion.
   */
  hipDeviceSynchronize();
}
